#include "hip/hip_runtime.h"
// cuda_kernels.cu
// The cuda device implementation and kernels reside here
//

#include "cuda_device.h"

#include <hip/hip_runtime.h>

#include <stdio.h>

#define CHECK_ERR do { if(err != hipSuccess) { \
	printf("CUDA call failed: %s(%d): %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
	return 0; } } while(0)

// scale kerne that executes on the CUDA device
__global__ void scale_d(double *x_in, double *y_in, int N, double scale) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<N) {
		x_in[idx] = x_in[idx] * scale;
		y_in[idx] = y_in[idx] * scale;
	}
}

__global__ void tmerc_sd(double *x_d, double *y_d, int N, double scale, double x0, double y0) {
	int i = threadIdx.x + blockDim.x*blockIdx.x;

	if(i < N) {
		double lambda = x_d[i] * 0.017453292519943295769236907684886;
		double phi    = y_d[i] * 0.017453292519943295769236907684886;

		double x, y, b, cosPhi, sinLambda, cosLambda;

		cosPhi = cos(phi);
		sinLambda = sin(lambda);
		cosLambda = cos(lambda);

		b = cosPhi * sinLambda;
		x = 0.5 * log((1.0 + b) / (1.0 - b));
		y = cosPhi * cosLambda / sqrt(1.0 - b * b);
		if (fabs(y) >= 1.0)
			y = 0.0;
		else
			y = acos(y);

		if (phi < 0.0) y = -y;

		x_d[i] = x0 + scale * x;
		y_d[i] = y0 + scale * y;
	}
}


static int 
prep_buffers(const double *x_in, const double *y_in, size_t size, double **x_d, double **y_d) {
	hipError_t err;

	err = hipMalloc(x_d, size * sizeof(double)); CHECK_ERR;
	err = hipMalloc(y_d, size * sizeof(double)); CHECK_ERR;

	err = hipMemcpy(*x_d, x_in, size * sizeof(double), hipMemcpyHostToDevice); CHECK_ERR;
	err = hipMemcpy(*y_d, y_in, size * sizeof(double), hipMemcpyHostToDevice); CHECK_ERR;

	return 1;
}

static int
unprep_buffers(double *x_d, double *y_d, size_t size, double *x, double *y) {
	hipError_t err;

	err = hipMemcpy(x, x_d, size * sizeof(double), hipMemcpyDeviceToHost); CHECK_ERR;
	err = hipMemcpy(y, y_d, size * sizeof(double), hipMemcpyDeviceToHost); CHECK_ERR;

	hipFree(x_d);
	hipFree(y_d);

	return 1;
}

int run_scale_transform(const double *x, const double *y, double *x_out, double *y_out,
		size_t n, double scale_with) {
	double *x_d, *y_d;

	if (!prep_buffers(x, y, n, &x_d, &y_d))
		return 1;


	int block_dim = 512;
	int block_count = (n + block_dim - 1) / block_dim;

	scale_d <<<block_count, block_dim>>>(x_d, y_d, (int)n, scale_with);
	hipError_t err = hipPeekAtLastError(); CHECK_ERR;

	unprep_buffers(x_d, y_d, n, x_out, y_out);

	return 0;
}

int run_tmerc_s_transform(const double *x, const double *y, double *x_out, double *y_out,
		size_t n, double scale, double x0, double y0) {
	double *x_d, *y_d;

	if (!prep_buffers(x, y, n, &x_d, &y_d))
		return 1;

	int block_dim = 512;
	int block_count = (n + block_dim - 1) / block_dim;

	tmerc_sd <<<block_count, block_dim>>>(x_d, y_d, (int)n, 6370997.0, 0.0, 0.0);

	unprep_buffers(x_d, y_d, n, x_out, y_out);
	
	return 0;
}
